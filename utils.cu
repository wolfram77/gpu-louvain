#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <vector>
#include <iostream>
#include <thrust/partition.h>
#include <fstream>
#include <getopt.h>
#include <sstream>
#include <algorithm>

using std::vector;
using std::string;
using std::fstream;
using std::istringstream;
using std::getline;
using std::sort;
using std::unique;




auto readEdges(char *filename) {
	fstream file;
	file.open(filename);
	string ln;
	do { getline(file, ln); }
	while (ln[0]=='%');
	istringstream ls(ln);
	int V, E;
	ls >> V >> V >> E;
	vector<vector<int>> edges(V);
	// read edges
  while (getline(file, ln)) {
    int u, v;
    ls = stringstream(ln);
    if (!(ls >> u >> v)) break;
		--u; --v;
		edges[u].push_back(v);
		edges[v].push_back(u);  // assume graph is symmetric
  }
	// remove duplicate edges
	for (int u=0; u<V; u++) {
		sort(edges[u].begin(), edges[u].end());
		auto it = unique(edges[u].begin(), edges[u].end());
		edges[u].resize(it - edges[u].begin());
	}
	return edges;
}



host_structures readInputData(char *fileName) {
	vector<vector<int>> edges = readEdges(filename);
	int V = edges.size(), E = 0;
	for (int u=0; u<V; u++)
		E += edges[u].size();
	int v1, v2;
	float w;
	host_structures hostStructures;
	hostStructures.originalV = V;
	hostStructures.V = V;
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.vertexCommunity, V * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.communityWeight, V * sizeof(float), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edgesIndex, (V + 1) * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.originalToCommunity, V * sizeof(int), hipHostMallocDefault));

	std::vector<std::vector<std::pair<int, float>>> neighbours(V);
	// TODO: here is assumption that graph is undirected
	int aux = E;
	for (int v1 = 0; v1 < V; v1++) {
		for (int v2 : edges[v1]) {
			float w = 1;  // assume weight = 1
			hostStructures.communityWeight[v1] += w;
			neighbours[v1].emplace_back(v2, w);
			// if (v1 != v2) {
			// 	E++;
			// 	hostStructures.communityWeight[v2] += w;
			// 	neighbours[v2].emplace_back(v1, w);
			// 	hostStructures.M += w;
			// }
			hostStructures.M += w;
		}
	}
	hostStructures.M /= 2;
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.edges, E * sizeof(int), hipHostMallocDefault));
	HANDLE_ERROR(hipHostAlloc((void**)&hostStructures.weights, E * sizeof(float), hipHostMallocDefault));
	hostStructures.E = E;
	int index = 0;
	for (int v = 0; v < V; v++) {
	hostStructures.edgesIndex[v] = index;
		for (auto & it : neighbours[v]) {
			hostStructures.edges[index]   = it.first;
			hostStructures.weights[index] = it.second;
			index++;
		}
	}
	hostStructures.edgesIndex[V] = E;
	file.close();
	return hostStructures;
}

void copyStructures(host_structures& hostStructures, device_structures& deviceStructures,
					aggregation_phase_structures& aggregationPhaseStructures) {
	// copying from deviceStructures to hostStructures
	int V = hostStructures.V, E = hostStructures.E;
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communityWeight, V * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edges, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.weights, E * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.edgesIndex, (V + 1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalToCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.vertexEdgesSum, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.newVertexCommunity, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.V, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.E, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.originalV, sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.communitySize, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&deviceStructures.partition, V * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&deviceStructures.toOwnCommunity, V * sizeof(int)));


	thrust::fill(thrust::device, deviceStructures.communitySize, deviceStructures.communitySize + V, 1);
	thrust::sequence(thrust::device, deviceStructures.vertexCommunity, deviceStructures.vertexCommunity + V, 0);
	thrust::sequence(thrust::device, deviceStructures.newVertexCommunity, deviceStructures.newVertexCommunity + V, 0);
	thrust::sequence(thrust::device, deviceStructures.originalToCommunity, deviceStructures.originalToCommunity + V, 0);

	HANDLE_ERROR(hipMemcpy(deviceStructures.communityWeight, hostStructures.communityWeight, V * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edges, hostStructures.edges, E * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.weights, hostStructures.weights, E * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.edgesIndex, hostStructures.edgesIndex, (V + 1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.V, &hostStructures.V, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.E, &hostStructures.E, sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(deviceStructures.originalV, &hostStructures.originalV, sizeof(int), hipMemcpyHostToDevice));

	// preparing aggregationPhaseStructures
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.communityDegree, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newID, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.edgePos, V * sizeof(int)));;
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.vertexStart, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.orderedVertices, V * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.edgeIndexToCurPos, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newEdges, E * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&aggregationPhaseStructures.newWeights, E * sizeof(float)));
}

void deleteStructures(host_structures& hostStructures, device_structures& deviceStructures,
					  aggregation_phase_structures& aggregationPhaseStructures) {
    HANDLE_ERROR(hipHostFree(hostStructures.vertexCommunity));
    HANDLE_ERROR(hipHostFree(hostStructures.communityWeight));
    HANDLE_ERROR(hipHostFree(hostStructures.edges));
    HANDLE_ERROR(hipHostFree(hostStructures.weights));
    HANDLE_ERROR(hipHostFree(hostStructures.edgesIndex));
    HANDLE_ERROR(hipHostFree(hostStructures.originalToCommunity));


	HANDLE_ERROR(hipFree(deviceStructures.originalV));
    HANDLE_ERROR(hipFree(deviceStructures.vertexCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.communityWeight));
	HANDLE_ERROR(hipFree(deviceStructures.edges));
	HANDLE_ERROR(hipFree(deviceStructures.weights));
	HANDLE_ERROR(hipFree(deviceStructures.edgesIndex));
	HANDLE_ERROR(hipFree(deviceStructures.originalToCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.vertexEdgesSum));
	HANDLE_ERROR(hipFree(deviceStructures.newVertexCommunity));
	HANDLE_ERROR(hipFree(deviceStructures.E));
	HANDLE_ERROR(hipFree(deviceStructures.V));
	HANDLE_ERROR(hipFree(deviceStructures.communitySize));
	HANDLE_ERROR(hipFree(deviceStructures.partition));
    HANDLE_ERROR(hipFree(deviceStructures.toOwnCommunity));

	HANDLE_ERROR(hipFree(aggregationPhaseStructures.communityDegree));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.newID));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.edgePos));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.vertexStart));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.orderedVertices));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.edgeIndexToCurPos));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.newEdges));
	HANDLE_ERROR(hipFree(aggregationPhaseStructures.newWeights));
}

int blocksNumber(int V, int threadsPerVertex) {
	return (V * threadsPerVertex + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
}

bool isPrime(int n) {
	for (int i = 2; i < sqrt(n) + 1; i++)
		if (n % i == 0)
			return false;
	return true;
}

int getPrime(int n) {
	do {
		n++;
	} while(!isPrime(n));
	return n;
}

void parseCommandLineArgs(int argc, char *argv[], float *minGain, bool *isVerbose, char **fileName) {
	bool isF, isG;
	char opt;
	while ((opt = getopt(argc, argv, "f:g:v")) != -1) {
		switch (opt) {
			case 'g':
				isG = true;
				*minGain = strtof(optarg, NULL);
				break;
			case 'v':
				*isVerbose = true;
				break;
			case 'f':
				isF = true;
				*fileName = optarg;
				break;
			default:
				printf("Usage: ./gpulouvain -f mtx-matrix-file -g min-gain [-v]\n");
				exit(1);
		}
	}
	if (!isF || !isG) {
		printf("Usage: ./gpulouvain -f mtx-matrix-file -g min-gain [-v]\n");
		exit(1);
	}
}
