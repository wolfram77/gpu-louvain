#include "utils.cuh"
#include "modularity_optimisation.cuh"
#include "community_aggregation.cuh"


int main(int argc, char *argv[]) {
	char *fileName;
	float minGain;
	bool isVerbose;
	parseCommandLineArgs(argc, argv, &minGain, &isVerbose, &fileName);
	printf("Using graph %s ...\n", fileName);

    auto hostStructures = readInputData(fileName);
		printf("Read input data\n");
    device_structures deviceStructures;
    aggregation_phase_structures aggregationPhaseStructures;

	printf("Copying structures to device memory ...\n");
    hipEvent_t start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));
	copyStructures(hostStructures, deviceStructures, aggregationPhaseStructures);
	initM(hostStructures);
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float memoryTime;
	HANDLE_ERROR(hipEventElapsedTime(&memoryTime, start, stop));
	printf("Memory time measured\n");

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));
	for (;;) {
		if (!optimiseModularity(minGain, deviceStructures, hostStructures))
			break;
		aggregateCommunities(deviceStructures, hostStructures, aggregationPhaseStructures);
	}
	int V;
	HANDLE_ERROR(hipMemcpy(&V, deviceStructures.V, sizeof(int), hipMemcpyDeviceToHost));
	printf("modularity: %f\n", calculateModularity(V, hostStructures.M, deviceStructures));
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float algorithmTime;
	HANDLE_ERROR(hipEventElapsedTime(&algorithmTime, start, stop));
	printf("algorithm_time: %f all_time: %f\n", algorithmTime, algorithmTime + memoryTime);
	if (isVerbose)
		printOriginalToCommunity(deviceStructures, hostStructures);
	deleteStructures(hostStructures, deviceStructures, aggregationPhaseStructures);
	printf("\n");
}
